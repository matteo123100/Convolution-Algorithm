#include "hip/hip_runtime.h"
﻿
// Codice Ottimizzato ed Asincrono
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cassert>
#include <vector>
#include <thread>
#include <fstream>

#define COLS 7680
#define ROWS 4320
#define KERNEL_SIZE 3
#define OUTPUT_ROWS (ROWS - KERNEL_SIZE + 1)
#define OUTPUT_COLS (COLS - KERNEL_SIZE + 1)

void CPU_convolutionRows(int, int);
void CPU_convolution(int);
int* CPU_inputMatrix = new int[ROWS * COLS];
int* CPU_kernel = new int[KERNEL_SIZE * KERNEL_SIZE];
int* CPU_outputMatrix = new int[(ROWS - KERNEL_SIZE + 1) * (COLS - KERNEL_SIZE + 1)];

void init_matrix(int*, int, int);
void stampaMatrix(int*, int, int);

__device__ int* d_input, * d_output;
__constant__  int d_kernel[KERNEL_SIZE * KERNEL_SIZE];

__global__ void convolution2DKernel(int* input, int* output)
{
    int IDrow = blockIdx.x * blockDim.x + threadIdx.x;
    int IDcol = blockIdx.y * blockDim.y + threadIdx.y;

    if ((IDcol < OUTPUT_COLS) && (IDrow < OUTPUT_ROWS))
    {
        int result = 0;
        for (int i = 0; i < KERNEL_SIZE; i++)
        {
            int passo = (i != 0) ? (ROWS - KERNEL_SIZE) * i : 0;
            for (int j = 0; j < KERNEL_SIZE; j++)
            {
                int indice = (IDrow + i) * COLS + IDcol + j; //int indice = (IDrow * inputCOLS) + i * kernelROWS + j + passo + IDcol;
                //if (IDrow == 1 && IDcol == 0)
                    //printf("Thread(%d, %d): | Input[%d]: %d | PER | kernel[%d]: %d |\n", IDrow, IDcol, indice, input[indice], i * KERNEL_SIZE + j, kernel[i * KERNEL_SIZE + j]);

                result += input[indice] * d_kernel[i * KERNEL_SIZE + j];
            }
        }
        //printf("Scrivo la out[%d]\n", IDrow * outputMatrixSize + IDcol);
        output[IDrow * OUTPUT_COLS + IDcol] = result;
    }
}



int main()
{
    int* input = new int[ROWS * COLS];
    int* kernel = new int[KERNEL_SIZE * KERNEL_SIZE];
    int* output = new int[OUTPUT_ROWS * OUTPUT_COLS];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    init_matrix(input, ROWS, COLS);
    init_matrix(kernel, KERNEL_SIZE, KERNEL_SIZE);

    printf("array:input \n");
    //stampaMatrix(input, ROWS, COLS);
    printf("\narray:kernel \n");
    //stampaMatrix(kernel, KERNEL_SIZE, KERNEL_SIZE);

    hipMalloc((void**)&d_input, ROWS * COLS * sizeof(int));
    //hipMalloc((void**)&d_kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(int));
    hipMalloc((void**)&d_output, OUTPUT_COLS * OUTPUT_ROWS * sizeof(int));

    //hipMemcpy(d_kernel, kernel, KERNEL_SIZE* KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(int));

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipMemcpyAsync(d_kernel, kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_input, input, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice, stream1);

    //hipMemcpy(d_input, input, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(d_kernel, kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);



    dim3 dimBlock(1, 256);
    int gridSizeX = (OUTPUT_COLS / dimBlock.x) != 0 ? std::ceil(((float)OUTPUT_COLS / (float)dimBlock.x)) : 1;
    int gridSizeY = (OUTPUT_ROWS / dimBlock.y) != 0 ? std::ceil(((float)OUTPUT_ROWS / (float)dimBlock.y)) : 1;

    dim3 dimGrid(gridSizeX, gridSizeY);

    //int gridSizeX = 1;
    //int gridSizeY = 1;
    //dim3 gridSize(dimGrid, dimGrid);

    printf("gridSizeX: %d , gridSizeY: %d\n", dimGrid, dimGrid);
    printf("outputMatrixSize: %dx%d\n", OUTPUT_ROWS, OUTPUT_COLS);
    hipMemcpy(d_input, input, (ROWS/2 + 2) * (COLS/2 + 2) * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    convolution2DKernel << <dimGrid, dimBlock >> > (d_input, d_output);
    //bool unaVOlta = true;
    for (int i = 0; i < ROWS; i += ROWS / 2)
    {
        for (int j = 0; j < COLS; j += COLS / 2)
        {
            if (i == 0 && j == 0)
                j = COLS / 2;

            int size;
            if(i == 0 && j == COLS / 2)
                size = (COLS / 2) * (ROWS / 2 + 2) * sizeof(int);
            else if(i == ROWS/2 && j == 0)
                size = (COLS / 2 + 2) * (ROWS / 2) * sizeof(int);
            else
                size = (COLS / 2) * (ROWS / 2) * sizeof(int);

            

            int offset = i * COLS + j;
            int offsetY = i;
            int offsetX = j;

            // Copy a sub-block of the input image to device
            hipMemcpyAsync(&d_input[offset], &input[offset], size, hipMemcpyHostToDevice, stream1);
            //if (unaVOlta)
            //{
            //    unaVOlta = false;

           // }
            // Launch convolution kernel on the sub-block
            convolution2DKernel << <dimGrid, dimBlock, 0, stream1 >> > (d_input, d_output);
            //convolution2DKernel << <gridSize, blockSize >> > (d_input, d_kernel, d_output);
            // Copy the result back to host
            hipMemcpyAsync(&output[offset], &d_output[offset], size, hipMemcpyDeviceToHost, stream1);
        }
    }



    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpyAsync(output, d_output, OUTPUT_ROWS * OUTPUT_COLS * sizeof(int), hipMemcpyDeviceToHost, stream2);
    // hipMemcpy(output, d_output, OUTPUT_ROWS * OUTPUT_COLS * sizeof(int), hipMemcpyDeviceToHost);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
    printf("\nTempo impiegato %f\n", elapsedTime);
    std::string s = "8k.txt";
    std::ofstream fout(s, std::ios::app);
    fout << elapsedTime << "\n";
    fout.close();

    printf("OUTPUT:\n");
    //stampaMatrix(output, OUTPUT_ROWS, OUTPUT_COLS);

    // Fine parte GPU

    // IMplementa la parte per CPU e poi fai il confronto


    return 0;
}

void init_matrix(int* m, int rows, int cols)
{
    int valore = 1;
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            m[cols * i + j] = valore;
            valore++;
        }
    }
}

void stampaMatrix(int* outputMatrix, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
            std::cout << outputMatrix[i * cols + j] << "\t";
        std::cout << std::endl;
    }
}

void CPU_convolutionRows(int startRow, int endRow) //DWORD WINAPI convolutionRows(LPVOID lpParam)//(int startRow, int endRow)
{
    //auto params = reinterpret_cast<std::pair<int, int>*>(lpParam);
    //int startRow = params->first;
    //int endRow = params->second;
    int outputColSize = COLS - KERNEL_SIZE + 1; // 1080 - 2 + 1 --> 1079
    int value;

    for (int row = startRow; row < endRow; row++)  // row � [0,1919]
    {
        for (int col = 0; col < outputColSize; col++) // col � [0,1079]
        {
            value = 0;
            for (int kRow = 0; kRow < KERNEL_SIZE; kRow++) // kRow € [0,1]
            {
                for (int kCol = 0; kCol < KERNEL_SIZE; kCol++) // kCol € [0,1]
                {
                    value += CPU_inputMatrix[(row + kRow) * COLS + (col + kCol)] * CPU_kernel[kRow * KERNEL_SIZE + kCol];
                }
            }
            CPU_outputMatrix[row * outputColSize + col] = value;
        }
    }
}

void CPU_convolution(int N_THREAD)
{
    int rowsPerThread = ROWS / N_THREAD;
    std::vector<std::thread> threads;
    for (int i = 0; i < N_THREAD; ++i)
    {
        int startRow = i * rowsPerThread;

        int endRow = (i == N_THREAD - 1) ? (ROWS - KERNEL_SIZE + 1) : (startRow + rowsPerThread);

        auto params = new std::pair<int, int>(startRow, endRow);
        //threadParams.push_back(*params);

        // Creazione del thread sospeso
        //HANDLE hThread = CreateThread(nullptr, 0, convolutionRows, params, CREATE_SUSPENDED, nullptr);
        //threadHandles.push_back(hThread);

        threads.emplace_back(CPU_convolutionRows, startRow, endRow);
    }

    //begin = steady_clock::now(); // ***** TIMER START
    for (auto& thread : threads)
    {
        thread.join();
    }
    //end = steady_clock::now();
}
